#include "qsort_cuda.cuh"

#include <cstdio>
#include <cstdlib>

void run_qsort(unsigned int* data, unsigned int nitems) {
  CHECK_CUDA_ERR(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

  int left = 0;
  int right = nitems - 1;
  std::fprintf(stdout, "Launching kernel on the GPU\n");
  cdp_simple_quicksort<<<1, 1>>>(data, left, right, 0);
  CHECK_CUDA_ERR(hipDeviceSynchronize());
}

void initialize_data(unsigned int* dst, unsigned int nitems) {
  srand(2047);
  for (unsigned i = 0; i < nitems; i++)
    dst[i] = rand() % nitems;
}

int main() {
  int num_items;
  std::fprintf(stdout, "Type in the number of items: ");
  std::fscanf(stdin, "%d", &num_items);

  int device_count = 0, device = -1;
  CHECK_CUDA_ERR(hipGetDeviceCount(&device_count));
  for (int i = 0; i < device_count; ++i) {
    hipDeviceProp_t properties;
    CHECK_CUDA_ERR(hipGetDeviceProperties(&properties, i));
    if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5)) {
      device = i;
      std::fprintf(stdout, "Running on GPU %d (%s)\n", i, properties.name);
      break;
    }
    std::fprintf(stdout, "GPU %d (%s) does not support CUDA Dynamic Parallelism\n", i, properties.name);
  }
  if (device == -1) {
    std::fprintf(stderr, "QSortCUDASimple requires GPU devices with compute SM 3.5 or higher.  Exiting...\n");
    exit(EXIT_FAILURE);
  }

  hipSetDevice(device);

  unsigned int *h_data = 0;
  unsigned int *d_data = 0;

  std::fprintf(stdout, "Initializing data:\n");
  h_data = (unsigned int *)malloc(num_items * sizeof(unsigned int));
  initialize_data(h_data, num_items);

  for (int i = 0; i < num_items; i++)
    std::fprintf(stdout, "Data [%u]: \n", h_data[i]);

  CHECK_CUDA_ERR(hipMalloc((void **)&d_data, num_items * sizeof(unsigned int)));
  CHECK_CUDA_ERR(hipMemcpy(d_data, h_data, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

  std::fprintf(stdout, "Running quicksort on %d elements\n", num_items);
  run_qsort(d_data, num_items);

  CHECK_CUDA_ERR(hipMemcpy(h_data, d_data, num_items * sizeof(unsigned int), hipMemcpyDeviceToHost));

  for (int i = 0; i < num_items; i++)
    std::fprintf(stdout, "Data [%u]: \n", h_data[i]);

  free(h_data);
  CHECK_CUDA_ERR(hipFree(d_data));

  exit(EXIT_SUCCESS);
}